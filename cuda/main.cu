#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <utility>
#include <chrono>
#include <iomanip>
#include <string>
#include <fstream>
#include <sstream>
#include <iostream>
#include <stdio.h>
#define BLOCK_SIZE 32
#define ROWS 2048
#define COLUMNS 2048
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t mulWithCuda(float* a, float* b, float* c, int N);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__global__ void simple_mul(
    float* X,
    float* Y,
    float* S,
    const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float res = 0;
    for (int k = 0; k < N; k++) {
        res += X[j * N + k] * Y[k * N + i];
    }
    S[j * N + i] = res;
}
__global__ void local_mul(float* X, float* Y, float* S,
    const int K) {
    int global_row = blockIdx.x * blockDim.x + threadIdx.x;
    int global_col = blockIdx.y * blockDim.y + threadIdx.y;
    int local_row = threadIdx.x;
    int local_col = threadIdx.y;

    __shared__ float localX[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float localY[BLOCK_SIZE][BLOCK_SIZE];
    float res = 0;
    for (int kg = 0; kg < K / BLOCK_SIZE; kg++) {
        int aid = global_col * K + (kg * BLOCK_SIZE + local_row);
        int bid = (kg * BLOCK_SIZE + local_col) * K + global_row;
        localX[local_col][local_row] = X[aid];
        localY[local_col][local_row] = Y[bid];
        __syncthreads();

        for (int i = 0; i < BLOCK_SIZE; i++) {
            res += localX[local_col][i] * localY[i][local_row];
        }
        __syncthreads();
    }

    S[global_col * K + global_row] = res;
}
int main()
{
    float* a = new float[ROWS * COLUMNS];
    int N = ROWS;
    for (size_t i = 0; i < ROWS; i++)
    {
        for (size_t j = 0; j < COLUMNS; j++)
        {
            a[i * COLUMNS + j] = rand() / 1e5;
            //a[i * COLUMNS + j] = 1;

        }
    }
    float* b = new float[ROWS * COLUMNS];
    for (size_t i = 0; i < ROWS; i++)
    {
        for (size_t j = 0; j < COLUMNS; j++)
        {
            b[i * COLUMNS + j] = rand() / 1e5;
            //b[i * COLUMNS + j] = 1;

        }
    }
    float* res = new float[ROWS * COLUMNS];
    for (size_t i = 0; i < ROWS; i++)
    {
        for (size_t j = 0; j < COLUMNS; j++)
        {
            res[i * COLUMNS + j] = 0;
        }
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = mulWithCuda(a, b, res, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
 /*   std::cout << "\nMatrix #RESULT: \n";*/
   /* for (size_t i = 0; i < ROWS; i++)
    {
        for (size_t j = 0; j < COLUMNS; j++)
        {
            std::cout << res[i * COLUMNS + j] << " ";
        }
        std::cout << "\n";
    }
   */
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
hipError_t mulWithCuda(float* a, float* b, float* c, int N)
{
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
  
    cudaStatus = hipMalloc((void**)&dev_c, ROWS * COLUMNS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, ROWS * COLUMNS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, ROWS * COLUMNS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, ROWS * COLUMNS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, ROWS * COLUMNS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    unsigned int grid_rows = (ROWS + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (COLUMNS + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // Launch a kernel on the GPU with one thread for each element.
    local_mul<<<dimGrid, dimBlock>>> (dev_a, dev_b, dev_c, N);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, ROWS * COLUMNS * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
